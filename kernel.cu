#include "hip/hip_runtime.h"
/*
 * http://github.com/dusty-nv/jetson-inference
 */
 
#include "cudaUtility.h"
#include <iostream>


// gpuPreImageNet
__global__ void gpuPreImageNet( float2 scale, float4* input, int iWidth, float* output, int oWidth, int oHeight )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int n = oWidth * oHeight;
	
	if( x >= oWidth || y >= oHeight )
		return;

	const int dx = ((float)x * scale.x);
	const int dy = ((float)y * scale.y);

	const float4 px  = input[ dy * iWidth + dx ];
	const float3 bgr = make_float3(px.z, px.y, px.x);
	
	output[n * 0 + y * oWidth + x] = bgr.x;
	output[n * 1 + y * oWidth + x] = bgr.y;
	output[n * 2 + y * oWidth + x] = bgr.z;
}


// cudaPreImageNet
hipError_t cudaPreImageNet( float4* input, size_t inputWidth, size_t inputHeight,
				         float* output, size_t outputWidth, size_t outputHeight )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 )
		return hipErrorInvalidValue;

	const float2 scale = make_float2( float(inputWidth) / float(outputWidth),
							    float(inputHeight) / float(outputHeight) );

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuPreImageNet<<<gridDim, blockDim>>>(scale, input, inputWidth, output, outputWidth, outputHeight);

	return CUDA(hipGetLastError());
}




// gpuPreImageNetMean
__global__ void gpuPreImageNetMean( float2 scale, float4* input, int iWidth, float* output, int oWidth, int oHeight, float3 mean_value )
{
	const int x = blockIdx.x * blockDim.x + threadIdx.x;
	const int y = blockIdx.y * blockDim.y + threadIdx.y;
	const int n = oWidth * oHeight;
	
	if( x >= oWidth || y >= oHeight )
		return;

	const int dx = ((float)x * scale.x);
	const int dy = ((float)y * scale.y);

	const float4 px  = input[ dy * iWidth + dx ];
	const float3 bgr = make_float3(px.z - mean_value.x, px.y - mean_value.y, px.x - mean_value.z);
	
	output[n * 0 + y * oWidth + x] = bgr.x;
	output[n * 1 + y * oWidth + x] = bgr.y;
	output[n * 2 + y * oWidth + x] = bgr.z;
}


// cudaPreImageNetMean
hipError_t cudaPreImageNetMean( float4* input, size_t inputWidth, size_t inputHeight,
				             float* output, size_t outputWidth, size_t outputHeight, const float3& mean_value )
{
	if( !input || !output )
		return hipErrorInvalidDevicePointer;

	if( inputWidth == 0 || outputWidth == 0 || inputHeight == 0 || outputHeight == 0 )
		return hipErrorInvalidValue;

	const float2 scale = make_float2( float(inputWidth) / float(outputWidth),
							    float(inputHeight) / float(outputHeight) );

	// launch kernel
	const dim3 blockDim(8, 8);
	const dim3 gridDim(iDivUp(outputWidth,blockDim.x), iDivUp(outputHeight,blockDim.y));

	gpuPreImageNetMean<<<gridDim, blockDim>>>(scale, input, inputWidth, output, outputWidth, outputHeight, mean_value);

	return CUDA(hipGetLastError());
}


__global__ void kernel_extract_roi(float* input, float* output, char* mean,
    const int input_w, const int output_w, const int output_h,
    const int in_plane_r, const int in_plane_g, const int in_plane_b,
    const int out_plane_r, const int out_plane_g, const int out_plane_b,
    const int bbox_x, const int bbox_y, const int bbox_w, const int bbox_h)
{
    uint x = blockIdx.x * blockDim.x + threadIdx.x;
    uint y = blockIdx.y * blockDim.y + threadIdx.y;

    if( x < output_w && y < output_h)
    {
        float r[2] = { float(x) * bbox_w / output_w + bbox_x,
                       float(y) * bbox_h / output_h + bbox_y };

        int   pos[4][2] = { { int(floor(r[0])), int(floor(r[1])) },
                            { int( ceil(r[0])), int(floor(r[1])) },
                            { int(floor(r[0])),  int(ceil(r[1])) },
                            { int( ceil(r[0])),  int(ceil(r[1])) } };

        float u = r[0]-floor(r[0]);
        float v = r[1]-floor(r[1]);

        float s[4] = { (1-u)*(1-v), u*(1-v), (1-u)*v, u*v };

        int map[4] = { pos[0][1]*input_w + pos[0][0], pos[1][1]*input_w + pos[1][0],
                       pos[2][1]*input_w + pos[2][0], pos[3][1]*input_w + pos[3][0]};

        int idx = y * output_w + x;
        output[idx+out_plane_r] = round( s[0]*input[map[0]+in_plane_r]
                                       + s[1]*input[map[1]+in_plane_r]
                                       + s[2]*input[map[2]+in_plane_r]
                                       + s[3]*input[map[3]+in_plane_r] );// float(mean[idx+out_plane_r]));
        output[idx+out_plane_g] = round( s[0]*input[map[0]+in_plane_g]
                                       + s[1]*input[map[1]+in_plane_g]
                                       + s[2]*input[map[2]+in_plane_g]
                                       + s[3]*input[map[3]+in_plane_g] );//float(mean[idx+out_plane_g]));
        output[idx+out_plane_b] = round( s[0]*input[map[0]+in_plane_b]
                                       + s[1]*input[map[1]+in_plane_b]
                                       + s[2]*input[map[2]+in_plane_b]
                                       + s[3]*input[map[3]+in_plane_b] );//float(mean[idx+out_plane_b]));
    }
}

void convertROI(float* input, float* output, char* mean, const int* srcSize, const int* dstSize, const int* roi, hipStream_t stream)
{
    int in_plane_r = 0;
    int in_plane_g = srcSize[1] * srcSize[2];
    int in_plane_b = srcSize[1] * srcSize[2] * 2;

    int out_plane_r = 0;
    int out_plane_g = dstSize[1] * dstSize[2];
    int out_plane_b = dstSize[1] * dstSize[2] * 2;

    int bbox_x = min(max(roi[0], 0), srcSize[2]-1);
    int bbox_y = min(max(roi[1], 0), srcSize[1]-1);
    int bbox_w = min(max(roi[2]-roi[0], 0), srcSize[2]-bbox_x-1 );
    int bbox_h = min(max(roi[3]-roi[1], 0), srcSize[1]-bbox_y-1 );

    dim3 dimBlock(32,32);
    dim3 dimGrid(dstSize[2]/dimBlock.x+1, dstSize[1]/dimBlock.y+1);

    std::cout << "ROI: " << bbox_x << " " << bbox_y << " " << bbox_w << " " << bbox_h << std::endl;

    kernel_extract_roi <<< dimGrid, dimBlock, 0, stream >>> (input, output, mean,
                       srcSize[2], dstSize[2], dstSize[1],
                       in_plane_r,   in_plane_g,  in_plane_b,
                       out_plane_r, out_plane_g, out_plane_b,
                       bbox_x, bbox_y, bbox_w, bbox_h);
}

